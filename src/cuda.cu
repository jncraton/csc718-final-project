#include "hip/hip_runtime.h"
#include "shared.h"

__global__ void update_from_gravity (Body * bodies, int * N) {
  double r2, new_mass;
  int i,j;
  double earth_density = earth_mass / ((4.0 / 3.0) * 3.14159 * earth_radius * earth_radius * earth_radius);

  i = blockIdx.x + 1;

  for (j = threadIdx.x; j < *N; j+=blockDim.x) {
    if (i != j && bodies[i].mass && bodies[j].mass) {
      // 12 Operations
      r2 = (
        (bodies[i].x-bodies[j].x) * (bodies[i].x-bodies[j].x) +
        (bodies[i].y-bodies[j].y) * (bodies[i].y-bodies[j].y) +
        (bodies[i].z-bodies[j].z) * (bodies[i].z-bodies[j].z)
      );

      double gravity_force = rsqrt(r2) * STEP_SIZE * G * bodies[j].mass * (1/r2);

      // 22 Operations (including 3 atomic)
      if (r2 > (bodies[i].radius + bodies[j].radius) * (bodies[i].radius + bodies[j].radius)) {
        atomicAdd(&bodies[i].dx, (bodies[j].x - bodies[i].x) * gravity_force);
        atomicAdd(&bodies[i].dy, (bodies[j].y - bodies[i].y) * gravity_force);
        atomicAdd(&bodies[i].dz, (bodies[j].z - bodies[i].z) * gravity_force);
      } else {
          new_mass = bodies[i].mass + bodies[j].mass;
          bodies[j].dx = (bodies[j].dx * bodies[j].mass + bodies[i].dx * bodies[i].mass) / new_mass;
          bodies[j].dy = (bodies[j].dy * bodies[j].mass + bodies[i].dy * bodies[i].mass) / new_mass;
          bodies[j].dz = (bodies[j].dz * bodies[j].mass + bodies[i].dz * bodies[i].mass) / new_mass;
          bodies[j].mass = new_mass;

          double volume = new_mass / earth_density;
           // Volume = (4/3) pi r^3
          // r^3 = volume * (3/4) / pi
          double r3 = volume * (3.0/4.0) / 3.14159;

          bodies[j].radius = cbrt(r3);
          bodies[i].mass = 0.0;
          bodies[i].radius = 0.0;
      }
    }
  }

}

__global__ void update_positions (Body * bodies, int * N) {
  // Update positions
  for (int i = threadIdx.x; i < *N; i+=blockDim.x) {
    bodies[i].x += bodies[i].dx * STEP_SIZE;
    bodies[i].y += bodies[i].dy * STEP_SIZE;
    bodies[i].z += bodies[i].dz * STEP_SIZE;
  }
}

// Nice GPU assertion code borrowed from:
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: Error code %d %s %s %d\n", code, hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void update(struct Body * bodies, int iterations) {
  Body * cuda_bodies;
  int * cuda_N;

  gpuErrchk(hipMalloc(&cuda_bodies, N * sizeof(Body)));
  gpuErrchk(hipMemcpy(cuda_bodies, bodies, N * sizeof(Body), hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc(&cuda_N, sizeof(int)));
  gpuErrchk(hipMemcpy(cuda_N, &N, sizeof(int), hipMemcpyHostToDevice));

  for (int i = 0; i < iterations; i++) {
    update_from_gravity<<<N-1,128>>>(cuda_bodies, cuda_N);
    update_positions<<<1,1024>>>(cuda_bodies, cuda_N);
  }

  gpuErrchk(hipMemcpy(bodies, cuda_bodies, N * sizeof(Body), hipMemcpyDeviceToHost));
  gpuErrchk(hipFree(cuda_bodies));
}

#include "main.h"
